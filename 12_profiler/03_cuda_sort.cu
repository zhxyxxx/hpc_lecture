#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hiprand.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

__global__ void fillBucket(unsigned *key, unsigned *bucket, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=n) return;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void scanBucket(unsigned *bucket, unsigned *offset, unsigned *buffer, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=range) return;
  grid_group grid = this_grid();
  offset[i] = bucket[i];
  for(int j=1; j<range; j<<=1) {
    buffer[i] = offset[i];
    grid.sync();
    if(i>=j) offset[i] += buffer[i-j];
    grid.sync();
  }
  offset[i] -= bucket[i];
}

__global__ void fillKey(unsigned *key, unsigned *bucket, unsigned *offset, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=range) return;
  int j = offset[i];
  for (; bucket[i]>0; bucket[i]--)
    key[j++] = i;
}

int main() {
  int n = 100000000;
  int m = 256;
  int range = 100000;
  unsigned *key, *bucket, *offset, *buffer;
  hipMallocManaged(&key, n*sizeof(unsigned));
  hipMallocManaged(&bucket, range*sizeof(unsigned));
  hipMallocManaged(&offset, range*sizeof(unsigned));
  hipMallocManaged(&buffer, range*sizeof(unsigned));
  for (int i=0; i<n; i++)
    key[i] = rand() % range;
  for (int i=0; i<range; i++)
    bucket[i] = 0;
  fillBucket<<<(n+m-1)/m,m>>>(key, bucket, n);
  void *args[] = {(void *)&bucket,  (void *)&offset, (void *)&buffer, (void*)&range};
  hipLaunchCooperativeKernel((void*)scanBucket, (range+m-1)/m, m, args);
  fillKey<<<(range+m-1)/m,m>>>(key, bucket, offset, range);
  hipDeviceSynchronize();
  for (int i=1; i<n; i++)
    assert(key[i] >= key[i-1]);
  hipFree(key);
  hipFree(bucket);
  hipFree(offset);
  hipFree(buffer);
}
