#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <hiprand.h>
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;

__global__ void fillBucket(unsigned *key, unsigned *bucket, int n, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=n) return;
  key[i] &= range - 1;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void scanBucket(unsigned *bucket, unsigned *offset, unsigned *buffer, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=range) return;
  grid_group grid = this_grid();
  offset[i] = bucket[i];
  for(int j=1; j<range; j<<=1) {
    buffer[i] = offset[i];
    grid.sync();
    if(i>=j) offset[i] += buffer[i-j];
    grid.sync();
  }
  offset[i] -= bucket[i];
}

__global__ void fillKey(unsigned *key, unsigned *bucket, unsigned *offset, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i>=range) return;
  int j = offset[i];
  for (; bucket[i]>0; bucket[i]--)
    key[j++] = i;
}

int main() {
  int n = 100000000;
  int m = 256;
  int range = 100000;
  unsigned *key, *bucket, *offset, *buffer;
  hipMallocManaged(&key, n*sizeof(unsigned));
  hipMallocManaged(&bucket, range*sizeof(unsigned));
  hipMallocManaged(&offset, range*sizeof(unsigned));
  hipMallocManaged(&buffer, range*sizeof(unsigned));
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); // Do we need this?
  hiprandGenerate(gen, key, n);
  hipDeviceSynchronize();
  for (int i=0; i<range; i++)
    bucket[i] = 0;
  fillBucket<<<(n+m-1)/m,m>>>(key, bucket, n, range);
  void *args[] = {(void *)&bucket,  (void *)&offset, (void *)&buffer, (void*)&range};
  hipLaunchCooperativeKernel((void*)scanBucket, (range+m-1)/m, m, args);
  fillKey<<<(range+m-1)/m,m>>>(key, bucket, offset, range);
  hipDeviceSynchronize();
  for (int i=1; i<n; i++)
    assert(key[i] >= key[i-1]);
  hipFree(key);
  hipFree(bucket);
  hipFree(offset);
  hipFree(buffer);
}
