#include "hip/hip_runtime.h"
#include <starpu.h>

void cpu_func(void *buffers[], void *) {
  float *a = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
  int N = (int)STARPU_VECTOR_GET_NX(buffers[0]);
  for(int i=0; i<N; i++)
    a[i] = i;
}

static __global__ void cuda_kernel(float *a) {
  int i = threadIdx.x;
  a[i] = -i;
}

void cuda_func(void *buffers[], void *) {
  float *a = (float *)STARPU_VECTOR_GET_PTR(buffers[0]);
  int N = STARPU_VECTOR_GET_NX(buffers[0]);
  cuda_kernel<<<1,N,0,starpu_cuda_get_local_stream()>>>(a);
  hipStreamSynchronize(starpu_cuda_get_local_stream());
}

int main() {
  const int N=8;
  float a[N];
  int ret = starpu_init(NULL);
  starpu_data_handle_t vector_handle;
  starpu_vector_data_register(&vector_handle,0,(uintptr_t)a,N,sizeof(float));
  starpu_codelet cl;
  starpu_codelet_init(&cl);
  cl.nbuffers = 1;
  cl.where = STARPU_CPU;
  cl.cpu_funcs[0] = cpu_func;
  starpu_task_insert(&cl,STARPU_RW,vector_handle,0);
  starpu_task_wait_for_all();
  for(int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  cl.where = STARPU_CUDA;
  cl.cuda_funcs[0] = cuda_func;
  starpu_task_insert(&cl,STARPU_RW,vector_handle,0);
  starpu_task_wait_for_all();
  starpu_data_unregister(vector_handle);
  for(int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  starpu_shutdown();
}
